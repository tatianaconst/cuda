#include "hip/hip_runtime.h"
#include "cuda_param.hpp"
#include "params.hpp"
#include "cuda_equation.hpp"



__constant__ static long ic;
__constant__ static long jc;
__constant__ static long kc;

__constant__ static int i_0;
__constant__ static int j_0;
__constant__ static int k_0;

__constant__ static float hx;
__constant__ static float hy;
__constant__ static float hz;
__constant__ static float ht;


void cuda_init(dvector &d_arrayPrev,
			   dvector &d_arrayCurr,
			   dvector &d_arrayNext,
			   long incsize)
{
	try {
       
		d_arrayPrev.resize(incsize);
		d_arrayCurr.resize(incsize);
		d_arrayNext.resize(incsize);
	}
    catch(...) {
        std::cerr << "CAUGHT AN EXCEPTION" << std::endl;
    }
    std::cout << "Resize OK" << std::endl;
}

void cuda_append(std::vector<hvector> &host, std::vector<dvector> &device, uint sz)
{
	host.push_back(hvector());
	host.back().resize(sz);
	device.push_back(dvector());
	device.back().resize(sz);
}

void cuda_copy_step(dvector &arrayPrev, dvector &arrayCurr, dvector &arrayNext)
{
	arrayPrev = arrayCurr;
	arrayCurr = arrayNext;
}



__device__
float phi(double x, double y, double z) 
{
  return sin(y) * cos(x - M_PI_2) * cos(z - M_PI_2);
}


__device__
float u(double x, double y, double z, double t) 
{
  return phi(x, y, z) * cos(t);
}


__device__
long index(uint i, uint j, uint k)
{
  return ((i + 1) * (jc + 2) + (j + 1)) * (kc + 2) + (k + 1);
}

__device__
long index2(uint j, uint k)
{
  return  (j + 1) * (kc + 2) + (k + 1);
}

__device__
float x(uint i)
{ 
  return (i_0 + i) * hx; 
}

__device__
float y(uint j)
{ 
  return (j_0 + j) * hy; 
}

__device__
float z(uint k)
{ 
  return (k_0 + k) * hz; 
}

__device__
float deltaTime(uint n) 
{ 
	return n * ht; 
}

__device__
float calculateIndex(uint i, uint j, uint k, 
					float *arrayCurr,
					float *arrayPrev) 
{
  long indexC = index(i, j, k);

  return 2 * arrayCurr[indexC] - arrayPrev[indexC] +
      ht * ht *
          ((arrayCurr[index(i - 1, j, k)] - 2 * arrayCurr[indexC] +
            arrayCurr[index(i + 1, j, k)]) /
               hx / hx +
           (arrayCurr[index(i, j - 1, k)] - 2 * arrayCurr[indexC] +
            arrayCurr[index(i, j + 1, k)]) /
               hy / hy +
           (arrayCurr[index(i, j, k - 1)] - 2 * arrayCurr[indexC] +
            arrayCurr[index(i, j, k + 1)]) /
               hz / hz);
}


struct i_j_k
{
	int i, j, k;

	__device__
	i_j_k (long offset)
	{
		long ij = offset / (kc + 2);
		i = ij / (jc + 2);
		j = ij % (jc + 2);
		k = offset % (kc + 2);
	}
};

struct j_k
{
	int j, k;

	__device__
	j_k (long offset)
	{
		j = offset / (kc + 2);
		// i = ij / (jc + 2);
		// j = ij % (jc + 2);
		k = offset % (kc + 2);
	}
};

struct i_j
{
	int i, j;

	__device__
	i_j (long offset)
	{
		i = offset / (jc + 2);
		// i = ij / (jc + 2);
		// j = ij % (jc + 2);
		j = offset % (jc + 2);
	}
};

struct i_k
{
	int i, k;

	__device__
	i_k (long offset)
	{
		i = offset / (kc + 2);
		// i = ij / (jc + 2);
		// j = ij % (jc + 2);
		k = offset % (kc + 2);
	}
};

// struct residual_functor
// {
// 	uint curr_step;
// 	float *arrayNext;

// 	residual_functor(uint step, float *array)
// 	:curr_step(step), arrayNext(array)
// 	{}

// 	__device__
// 	float operator()(const float &acc, const long &offset)
// 	{
// 		i_j_k index(offset);
// 		if (index.i == 0 || index.i == ic + 1 ||
// 			index.j == 0 || index.j == jc + 1 ||
// 			index.k == 0 || index.k == kc + 1)
// 			return acc;

// 		float aSol = u(x(index.i), y(index.j), z(index.k), 
// 					   deltaTime(curr_step));
// 		float residual = aSol - arrayNext[offset];
// 		return acc + std::abs(residual);		
// 	}
// };



// __host__
// float cuda_residual(uint curr_step, dvector arrayNext)
// {
// 	thrust::counting_iterator<int> it(0);
// 	return thrust::reduce(it, it + arrayNext.size(),
// 								  float(0.0),
// 						   		  residual_functor(curr_step, arrayNext.data().get())
// 						   		 );
// }

// __device__
// float sumRes;

struct residual_functor
{
	uint curr_step;
	float *arrayNext;

	residual_functor(uint step, float *array)
	:curr_step(step), arrayNext(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k index(offset);
		if (index.i == 0 || index.i == ic + 1 ||
			index.j == 0 || index.j == jc + 1 ||
			index.k == 0 || index.k == kc + 1)
			return 0.0;

		float aSol = u(x(index.i), y(index.j), z(index.k), 
					   deltaTime(curr_step));
		float residual = aSol - arrayNext[offset];
		return std::abs(residual);		
	}
};

__host__
float cuda_residual(uint curr_step, dvector arrayNext)
{
	dvector resVec(arrayNext.size());
	thrust::counting_iterator<int> it(0);
	thrust::transform(it, it + resVec.size(), resVec.begin(),
					  residual_functor(curr_step, arrayNext.data().get()));
	return thrust::reduce(resVec.begin(), resVec.end()) / resVec.size();
	// return thrust::inclusive_scan(arrayNext.begin(), arrayNext.end(),
	// 							  float(0.0),
	// 					   		  residual_functor(curr_step)
	// 					   		 );
}



struct initPrev_functor
{
	__device__
	float operator()(long offset) 
	{
		i_j_k index(offset);
		return phi(x(index.i), y(index.j), z(index.k));
	}
};

__host__
void cuda_initPrev(dvector &arrayPrev)
{
	thrust::counting_iterator<int> it(0);
	thrust::transform(it, it + arrayPrev.size(), 
					  arrayPrev.begin(), initPrev_functor());
}


struct initCurr_functor
{
	float *arrayPrev;

	__host__
	initCurr_functor(float *array)
	:arrayPrev(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k index(offset);
		return arrayPrev[offset] + ht * ht / 2 * 
			   (-phi(x(index.i), y(index.j), z(index.k)));
	}
};

__host__
void cuda_initCurr(dvector &arrayPrev, dvector &arrayCurr)
{
	thrust::counting_iterator<int> it;
	thrust::transform(it, it + arrayCurr.size(), arrayCurr.begin(),
					  initCurr_functor(arrayPrev.data().get()));
}

struct calculateIndex_functor
{
	float *arrayCurr;
	float *arrayPrev;

	__host__
	calculateIndex_functor(float *array, float *arrayP)
	:arrayCurr(array), arrayPrev(arrayP)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k index(offset);
		return calculateIndex(index.i, index.j, index.k, arrayCurr, arrayPrev);
	}
};

struct calculateIndexDir_functor
{
	float *arrayCurr;
	float *arrayPrev;

	__host__
	calculateIndexDir_functor(float *array, float *arrayP)
	:arrayCurr(array), arrayPrev(arrayP)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k idx(offset);
		if (idx.i == 0 || idx.i == ic - 1 || 
			idx.j == 0 || idx.j == jc - 1 || 
			idx.k == 0 || idx.k == kc - 1)
			return calculateIndex(idx.i, idx.j, idx.k, arrayCurr, arrayPrev);
	}
};


__host__
void cuda_calculateIndex(dvector &d_arrayNext, dvector &d_arrayCurr, 
						 dvector &d_arrayPrev)
{
	thrust::counting_iterator<int> it;
	thrust::transform(it, it + d_arrayNext.size(), d_arrayNext.begin(),
					  calculateIndex_functor(d_arrayCurr.data().get(),
					  						 d_arrayPrev.data().get()));
}

void cuda_calculateDir(dvector &d_arrayNext, dvector &d_arrayCurr, 
						 dvector &d_arrayPrev)
{
	thrust::counting_iterator<int> it;
	thrust::transform(it, it + d_arrayNext.size(), d_arrayNext.begin(),
					  calculateIndexDir_functor(d_arrayCurr.data().get(),
					  						 d_arrayPrev.data().get()));
}


__device__
struct edgeX_send_functor
{
	int fix_i;
	float *arrayCurr;

	__host__
	edgeX_send_functor(int _i, float *array)
	:fix_i(_i), arrayCurr(array)
	{}

	__device__
	float operator()(long offset)
	{
		j_k idx(offset);
		return arrayCurr[index(fix_i, idx.j, idx.k)];
	}

};

__device__
struct edgeY_send_functor
{
	int fix_j;
	float *arrayCurr;

	__host__
	edgeY_send_functor(int j, float *array)
	:fix_j(j), arrayCurr(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_k idx(offset);
		return arrayCurr[index(idx.i, fix_j, idx.k)];
	}

};

__device__
struct edgeZ_send_functor
{
	int fix_k;
	float *arrayCurr;

	__host__
	edgeZ_send_functor(int k, float *array)
	:fix_k(k), arrayCurr(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_j idx(offset);
		return arrayCurr[index(idx.i, idx.j, fix_k)];
	}

};

__device__
struct edgeX_recv_functor
{
	int fix_i;
	float *v;

	__host__
	edgeX_recv_functor(int _i, float *array)
	:fix_i(_i), v(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k idx(offset);
		if (idx.i == fix_i)
			return v[index2(idx.j, idx.k)];
	}
};

__device__
struct edgeY_recv_functor
{
	int fix_j;
	float *v;

	__host__
	edgeY_recv_functor(int _j, float *array)
	:fix_j(_j), v(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k idx(offset);
		if (idx.j == fix_j)
			return v[index2(idx.i, idx.k)];
	}
};

__device__
struct edgeZ_recv_functor
{
	int fix_k;
	float *v;

	__host__
	edgeZ_recv_functor(int _k, float *array)
	:fix_k(_k), v(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k idx(offset);
		if (idx.k == fix_k)
			return v[index2(idx.i, idx.j)];
	}
};


void cuda_edgeX(ExchangeDir cdir, dvector &v, uint id, bool recv, 
				dvector d_arrayNext, dvector d_arrayCurr) 
{
  // ExchangeDir cdir = requests.iv[id];
  // dvector &v = requests.device[id];
  int i;
  switch (cdir) {
  case plus_x: {
    i = recv ? ic : ic - 1;
    break;
  }
  case minus_x: {
    i = recv ? -1 : 0;
    break;
  }
  case period_plus_x: {
    i = recv ? ic : ic - 1;
    break;
  }
  case period_minus_x: {
    i = recv ? 0 : 1;
    break;
  }
  }
  // dvector &a =
  //     (((cdir == period_minus_x) && recv) ? d_arrayNext : d_arrayCurr);
  // for (uint j = 0; j < jc; ++j) {
  //   for (uint k = 0; k < kc; ++k) {
  //     if (!recv)
  //       v[offset++] = d_arrayCurr[index(i, j, k)];
  //     else
  //       d_arrayCurr[index(i, j, k)] = v[offset++];
  //   }
  // }
  	thrust::counting_iterator<int> it;

	if (!recv) {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeX_send_functor(i, d_arrayCurr.data().get()));
	    // v[offset++] = d_arrayCurr[index(i, j, k)];
	}
	else {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeX_recv_functor(i, d_arrayCurr.data().get()));
		//d_arrayCurr[index(i, j, k)] = v[offset++];
	}
}

void cuda_edgeY(ExchangeDir cdir, dvector &v, uint id, bool recv, 
				dvector d_arrayNext, dvector d_arrayCurr) {
  // ExchangeDir cdir = requests.iv[id];
  // thrust::host_vector<float> &v = requests.host[id];
  int j;
  switch (cdir) {
  case plus_y: {
    j = recv ? jc : jc - 1;
    break;
  }
  case minus_y: {
    j = recv ? -1 : 0;
    break;
  }
  }

  // for (uint i = 0; i < ic; ++i) {
  //   for (uint k = 0; k < kc; ++k) {
  //     inRange(offset, 0, v.size());
  //     inRange(index(i, j, k), 0, d_arrayCurr.size());
  //     if (!recv)
  //       v[offset++] =d_arrayCurr[index(i, j, k)];
  //     // copy_send(v, arrayCurr, i, j, k, offset++);
  //     else
  //       d_arrayCurr[index(i, j, k)] = v[offset++];
  //     // copy_recv(v, arrayCurr, i, j, k, offset++);
  //   }
  // }

  thrust::counting_iterator<int> it;

	if (!recv) {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeY_send_functor(j, d_arrayCurr.data().get()));
	    // v[offset++] = d_arrayCurr[index(i, j, k)];
	}
	else {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeY_recv_functor(j, d_arrayCurr.data().get()));
		//d_arrayCurr[index(i, j, k)] = v[offset++];
	}
}

void cuda_edgeZ(ExchangeDir cdir, dvector &v, uint id, bool recv, 
				dvector d_arrayNext, dvector d_arrayCurr) {
  // ExchangeDir cdir = requests.iv[id];
  // std::vector<float> &v = requests.host[id];
  int k;
  switch (cdir) {
  case plus_z: {
    k = recv ? kc : kc - 1;
    break;
  }
  case minus_z: {
    k = recv ? -1 : 0;
    break;
  }
  case period_plus_z: {
    k = recv ? kc : kc - 1;
    break;
  }
  case period_minus_z: {
    k = recv ? 0 : 1;
    break;
  }
  }
  // std::vector<float> &a =
  //     (((cdir == period_minus_z) && recv) ? arrayNext : arrayCurr);
  // for (uint i = 0; i < ic; ++i) {
  //   for (uint j = 0; j < jc; ++j) {
  //     inRange(offset, 0, v.size());
  //     inRange(index(i, j, k), 0, arrayCurr.size());
  //     if (!recv)
  //       v[offset++] = arrayCurr[index(i, j, k)];
  //     else
  //       arrayCurr[index(i, j, k)] = v[offset++];
  //   }
  // }
    thrust::counting_iterator<int> it;

	if (!recv) {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeZ_send_functor(k, d_arrayCurr.data().get()));
	    // v[offset++] = d_arrayCurr[index(i, j, k)];
	}
	else {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeZ_recv_functor(k, d_arrayCurr.data().get()));
		//d_arrayCurr[index(i, j, k)] = v[offset++];
	}
}


