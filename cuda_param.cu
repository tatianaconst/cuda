#include "hip/hip_runtime.h"
#include "cuda_param.hpp"
#include "params.hpp"
#include "cuda_equation.hpp"



__constant__ static long ic;
__constant__ static long jc;
__constant__ static long kc;

__constant__ static int i_0;
__constant__ static int j_0;
__constant__ static int k_0;

__constant__ static float hx;
__constant__ static float hy;
__constant__ static float hz;
__constant__ static float ht;

__constant__ static long incsize;


void cuda_init(dvector &d_arrayPrev,
			   dvector &d_arrayCurr,
			   dvector &d_arrayNext,
			   long _incsize, long _ic, long _jc, long _kc,
			   int _i_0, int _j_0, int _k_0,
			   float _hx, float _hy, float _hz, float _ht)
{
	hipMemcpyToSymbol(HIP_SYMBOL(incsize), &_incsize, sizeof(incsize), 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(ic), &_ic, sizeof(ic), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(jc), &_jc, sizeof(jc), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(kc), &_kc, sizeof(kc), 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(i_0), &_i_0, sizeof(i_0), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(j_0), &_j_0, sizeof(j_0), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(k_0), &_k_0, sizeof(k_0), 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(hx), &_hx, sizeof(hx), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(hy), &_hy, sizeof(hy), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(hz), &_hz, sizeof(hz), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(ht), &_ht, sizeof(ht), 0, hipMemcpyHostToDevice);


	try {
       
		d_arrayPrev.resize(_incsize);
		d_arrayCurr.resize(_incsize);
		d_arrayNext.resize(_incsize);
	}
    catch(...) {
        std::cerr << "CAUGHT AN EXCEPTION" << std::endl;
    }
    std::cout << "Resize OK" << std::endl;
}

void cuda_append(std::vector<hvector> &host, std::vector<dvector> &device, uint sz)
{
	host.push_back(hvector());
	host.back().resize(sz);
	device.push_back(dvector());
	device.back().resize(sz);
}

void cuda_copy_step(dvector &arrayPrev, dvector &arrayCurr, dvector &arrayNext)
{
	arrayPrev = arrayCurr;
	arrayCurr = arrayNext;
}



__device__
float phi(double x, double y, double z) 
{
  return sin(y) * cos(x - M_PI_2) * cos(z - M_PI_2);
}


__device__
float u(double x, double y, double z, double t) 
{
  return phi(x, y, z) * cos(t);
}


__device__
long index(uint i, uint j, uint k)
{
	return (long(i) * (jc + 2) + j) * (kc + 2) + k;
}

__device__
long index2(uint j, uint k)
{
  return  (j + 1) * (kc + 2) + (k + 1);
}

__device__
float x(uint i)
{ 
  return (i_0 + i) * hx; 
}

__device__
float y(uint j)
{ 
  return (j_0 + j) * hy; 
}

__device__
float z(uint k)
{ 
  return (k_0 + k) * hz; 
}

__device__
float deltaTime(uint n) 
{ 
	return n * ht; 
}

__device__
float calculateIndex(uint i, uint j, uint k, 
					float *arrayCurr,
					float *arrayPrev) 
{
  long indexC = index(i, j, k);

  return 2 * arrayCurr[indexC] - arrayPrev[indexC] +
      ht * ht *
          ((arrayCurr[index(i - 1, j, k)] - 2 * arrayCurr[indexC] +
            arrayCurr[index(i + 1, j, k)]) /
               hx / hx +
           (arrayCurr[index(i, j - 1, k)] - 2 * arrayCurr[indexC] +
            arrayCurr[index(i, j + 1, k)]) /
               hy / hy +
           (arrayCurr[index(i, j, k - 1)] - 2 * arrayCurr[indexC] +
            arrayCurr[index(i, j, k + 1)]) /
               hz / hz);
}


struct i_j_k
{
	int i, j, k;

	__device__
	i_j_k (long offset)
	{
		long ij = offset / (kc + 2);
		i = ij / (jc + 2);
		j = ij % (jc + 2);
		k = offset % (kc + 2);
	}
};

struct j_k
{
	int j, k;

	__device__
	j_k (long offset)
	{
		j = offset / (kc + 2);
		// i = ij / (jc + 2);
		// j = ij % (jc + 2);
		k = offset % (kc + 2);
	}
};

struct i_j
{
	int i, j;

	__device__
	i_j (long offset)
	{
		i = offset / (jc + 2);
		// i = ij / (jc + 2);
		// j = ij % (jc + 2);
		j = offset % (jc + 2);
	}
};

struct i_k
{
	int i, k;

	__device__
	i_k (long offset)
	{
		i = offset / (kc + 2);
		// i = ij / (jc + 2);
		// j = ij % (jc + 2);
		k = offset % (kc + 2);
	}
};

struct residual_functor
{
	uint curr_step;
	float *arrayNext;

	residual_functor(uint step, float *array)
	:curr_step(step), arrayNext(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k index(offset);
		if (index.i == 0 || index.i == ic + 1 ||
			index.j == 0 || index.j == jc + 1 ||
			index.k == 0 || index.k == kc + 1)
			return 0.0;

		float aSol = u(x(index.i), y(index.j), z(index.k), 
					   deltaTime(curr_step));
		float residual = aSol - arrayNext[offset];
		return std::abs(residual);		
	}
};

__host__
float cuda_residual(uint curr_step, dvector arrayNext)
{
	dvector resVec(arrayNext.size());
	thrust::counting_iterator<int> it(0);
	thrust::transform(it, it + resVec.size(), resVec.begin(),
					  residual_functor(curr_step, arrayNext.data().get()));
	return thrust::reduce(resVec.begin(), resVec.end()) / resVec.size();
	// return thrust::inclusive_scan(arrayNext.begin(), arrayNext.end(),
	// 							  float(0.0),
	// 					   		  residual_functor(curr_step)
	// 					   		 );
}



struct initPrev_functor
{
	__device__
	float operator()(long offset) 
	{
		i_j_k index(offset);
		return phi(x(index.i), y(index.j), z(index.k));
	}
};

__host__
void cuda_initPrev(dvector &arrayPrev)
{
	thrust::counting_iterator<int> it(0);
	thrust::transform(it, it + arrayPrev.size(), 
					  arrayPrev.begin(), initPrev_functor());
}


struct initCurr_functor
{
	float *arrayPrev;

	__host__
	initCurr_functor(float *array)
	:arrayPrev(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k index(offset);
		return arrayPrev[offset] + ht * ht / 2 * 
			   (-phi(x(index.i), y(index.j), z(index.k)));
	}
};

__host__
void cuda_initCurr(dvector &arrayPrev, dvector &arrayCurr)
{
	thrust::counting_iterator<int> it;
	thrust::transform(it, it + arrayCurr.size(), arrayCurr.begin(),
					  initCurr_functor(arrayPrev.data().get()));
}

struct calculateIndex_functor
{
	float *arrayCurr;
	float *arrayPrev;
	float *arrayNext;

	__host__
	calculateIndex_functor(float *array, float *arrayP, float *arrayN)
	:arrayCurr(array), arrayPrev(arrayP), arrayNext(arrayN)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k idx(offset);
		if (idx.i > 1 && idx.i < ic - 2 && 
			idx.j > 1 && idx.j < jc - 2 && 
			idx.k > 1 && idx.k < kc - 2)
			return calculateIndex(idx.i, idx.j, idx.k, arrayCurr, arrayPrev);
		else
			return arrayNext[offset];
	}
};

struct calculateIndexDir_functor
{
	float *arrayCurr;
	float *arrayPrev;
	float *arrayNext;


	__host__
	calculateIndexDir_functor(float *array, float *arrayP, float *arrayN)
	:arrayCurr(array), arrayPrev(arrayP), arrayNext(arrayN)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k idx(offset);
		if (idx.i == 1 || idx.i == ic - 2 || 
			idx.j == 1 || idx.j == jc - 2 || 
			idx.k == 1 || idx.k == kc - 2)
			return calculateIndex(idx.i, idx.j, idx.k, arrayCurr, arrayPrev);
		else return arrayNext[offset];
	}
};


__host__
void cuda_calculateIndex(dvector &d_arrayNext, dvector &d_arrayCurr, 
						 dvector &d_arrayPrev)
{
	thrust::counting_iterator<int> it;
	thrust::transform(it, it + d_arrayNext.size(), d_arrayNext.begin(),
					  calculateIndex_functor(d_arrayCurr.data().get(),
					  						 d_arrayPrev.data().get(),
					  						 d_arrayNext.data().get()));
}

void cuda_calculateDir(dvector &d_arrayNext, dvector &d_arrayCurr, 
						 dvector &d_arrayPrev)
{
	thrust::counting_iterator<int> it;
	thrust::transform(it, it + d_arrayNext.size(), d_arrayNext.begin(),
					  calculateIndexDir_functor(d_arrayCurr.data().get(),
					  						 d_arrayPrev.data().get(), d_arrayNext.data().get()));
}

struct edgeX_send_functor
{
	int fix_i;
	float *arrayCurr;

	__host__
	edgeX_send_functor(int _i, float *array)
	:fix_i(_i), arrayCurr(array)
	{}

	__device__
	float operator()(long offset)
	{
		j_k idx(offset);
		return arrayCurr[index(fix_i, idx.j, idx.k)];
	}

};

struct edgeY_send_functor
{
	int fix_j;
	float *arrayCurr;

	__host__
	edgeY_send_functor(int j, float *array)
	:fix_j(j), arrayCurr(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_k idx(offset);
		return arrayCurr[index(idx.i, fix_j, idx.k)];
	}

};

struct edgeZ_send_functor
{
	int fix_k;
	float *arrayCurr;

	__host__
	edgeZ_send_functor(int k, float *array)
	:fix_k(k), arrayCurr(array)
	{}

	__device__
	float operator()(long offset)
	{
		i_j idx(offset);
		return arrayCurr[index(idx.i, idx.j, fix_k)];
	}

};

struct edgeX_recv_functor
{
	int fix_i;
	float *v;
	float *arrayCurr;

	__host__
	edgeX_recv_functor(int _i, float *array, float *arrayC)
	:fix_i(_i), v(array), arrayCurr(arrayC)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k idx(offset);
		if (idx.i == fix_i)
			return v[index2(idx.j, idx.k)];
		else 
			return arrayCurr[offset];
	}
};

struct edgeY_recv_functor
{
	int fix_j;
	float *v;
	float *def;

	__host__
	edgeY_recv_functor(int _j, float *array, float *d)
	:fix_j(_j), v(array), def(d)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k idx(offset);
		if (idx.j == fix_j)
			return v[index2(idx.i, idx.k)];
		else
			return def[offset];
	}
};

struct edgeZ_recv_functor
{
	int fix_k;
	float *v;
	float *def;

	__host__
	edgeZ_recv_functor(int _k, float *array, float *d)
	:fix_k(_k), v(array), def(d)
	{}

	__device__
	float operator()(long offset)
	{
		i_j_k idx(offset);
		if (idx.k == fix_k)
			return v[index2(idx.i, idx.j)];
		else
			return def[offset];
	}
};


void cuda_edgeX(ExchangeDir cdir, dvector &v, uint id, bool recv, 
				dvector &d_arrayNext, dvector &d_arrayCurr, long ic, long jc, long kc) 
{
  // ExchangeDir cdir = requests.iv[id];
  // dvector &v = requests.device[id];
  int i;
  switch (cdir) {
  case plus_x: {
    i = recv ? ic - 1 : ic - 2;
    break;
  }
  case minus_x: {
    i = recv ? 0 : 1;
    break;
  }
  case period_plus_x: {
    i = recv ? ic - 1 : ic - 2;
    break;
  }
  case period_minus_x: {
    i = recv ? 1 : 2;
    break;
  }
  }

  	thrust::counting_iterator<int> it;

	if (!recv) {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeX_send_functor(i, d_arrayCurr.data().get()));
	    // v[offset++] = d_arrayCurr[index(i, j, k)];
	}
	else {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeX_recv_functor(i, d_arrayCurr.data().get(), d_arrayCurr.data().get()));
		//d_arrayCurr[index(i, j, k)] = v[offset++];
	}
}

void cuda_edgeY(ExchangeDir cdir, dvector &v, uint id, bool recv, 
				dvector &d_arrayNext, dvector &d_arrayCurr, long jc) {
  // ExchangeDir cdir = requests.iv[id];
  // thrust::host_vector<float> &v = requests.host[id];
  int j;
  switch (cdir) {
  case plus_y: {
    j = recv ? jc - 1 : jc - 2;
    break;
  }
  case minus_y: {
    j = recv ? 0 : 1;
    break;
  }
  }

  thrust::counting_iterator<int> it;

	if (!recv) {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeY_send_functor(j, d_arrayCurr.data().get()));
	    // v[offset++] = d_arrayCurr[index(i, j, k)];
	}
	else {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeY_recv_functor(j, d_arrayCurr.data().get(), d_arrayCurr.data().get()));
		//d_arrayCurr[index(i, j, k)] = v[offset++];
	}
}

void cuda_edgeZ(ExchangeDir cdir, dvector &v, uint id, bool recv, 
				dvector &d_arrayNext, dvector &d_arrayCurr, long ic, long jc, long kc) {
  // ExchangeDir cdir = requests.iv[id];
  // std::vector<float> &v = requests.host[id];
  int k;
  switch (cdir) {
  case plus_z: {
    k = recv ? kc - 1 : kc - 2;
    break;
  }
  case minus_z: {
    k = recv ? 0 : 1;
    break;
  }
  case period_plus_z: {
    k = recv ? kc - 1 : kc - 2;
    break;
  }
  case period_minus_z: {
    k = recv ? 1 : 2;
    break;
  }
  }
    thrust::counting_iterator<int> it;

	if (!recv) {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeZ_send_functor(k, d_arrayCurr.data().get()));
	    // v[offset++] = d_arrayCurr[index(i, j, k)];
	}
	else {
		thrust::transform(it, it + v.size(), v.begin(),
					  edgeZ_recv_functor(k, d_arrayCurr.data().get(), d_arrayCurr.data().get()));
		//d_arrayCurr[index(i, j, k)] = v[offset++];
	}
}


