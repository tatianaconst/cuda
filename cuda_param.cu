#include "hip/hip_runtime.h"
#include "cuda_equation.hpp"
#include "cuda_param.hpp"
#include "params.hpp"

#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>

__device__ long c;
__constant__ static long ic;
__constant__ static long jc;
__constant__ static long kc;

__constant__ static int i_0;
__constant__ static int j_0;
__constant__ static int k_0;

__constant__ static double hx;
__constant__ static double hy;
__constant__ static double hz;
__constant__ static double ht;

__constant__ static long incsize;

void cuda_init(dvector& d_arrayPrev,
               dvector& d_arrayCurr,
               dvector& d_arrayNext,
               long _incsize,
               long _ic,
               long _jc,
               long _kc,
               int _i_0,
               int _j_0,
               int _k_0,
               double _hx,
               double _hy,
               double _hz,
               double _ht) {
  hipMemcpyToSymbol(HIP_SYMBOL(incsize), &_incsize, sizeof(incsize), 0,
                     hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(ic), &_ic, sizeof(ic), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(jc), &_jc, sizeof(jc), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(kc), &_kc, sizeof(kc), 0, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(i_0), &_i_0, sizeof(i_0), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(j_0), &_j_0, sizeof(j_0), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(k_0), &_k_0, sizeof(k_0), 0, hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(hx), &_hx, sizeof(hx), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(hy), &_hy, sizeof(hy), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(hz), &_hz, sizeof(hz), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(ht), &_ht, sizeof(ht), 0, hipMemcpyHostToDevice);

  try {
    d_arrayPrev.resize(_incsize);
    d_arrayCurr.resize(_incsize);
    d_arrayNext.resize(_incsize);
  } catch (...) {
    std::cerr << "CAUGHT AN EXCEPTION" << std::endl;
  }
}

long cuda_counter() {return c;}

void cuda_prev_to_next(dvector &arrayPrev, dvector &arrayNext)
{
  arrayNext = arrayPrev;
}

void cuda_append(std::vector<hvector>& host,
                 std::vector<dvector>& device,
                 uint sz) {
  host.push_back(hvector());
  host.back().resize(sz);
  device.push_back(dvector());
  device.back().resize(sz);
}

void cuda_copy_step(dvector& arrayPrev,
                    dvector& arrayCurr,
                    dvector& arrayNext) {
  arrayPrev = arrayCurr;
  arrayCurr = arrayNext;
}

__device__ double phi(double x, double y, double z) {
  return sin(y) * cos(x - M_PI_2) * cos(z - M_PI_2);
}

__device__ double u(double x, double y, double z, double t) {
  return phi(x, y, z) * cos(t);
}

__device__ long index(uint i, uint j, uint k) {
  return (long(i) * (jc + 2) + j) * (kc + 2) + k;
}

__device__ long index2(uint j, uint k) {
  return (j + 1) * (kc + 2) + (k + 1);
}

__device__ double x(uint i) {
  return (i_0 + i) * hx;
}

__device__ double y(uint j) {
  return (j_0 + j) * hy;
}

__device__ double z(uint k) {
  return (k_0 + k) * hz;
}

__device__ double deltaTime(uint n) {
  return n * ht;
}

__device__ double calculateIndex(uint i,
                                 uint j,
                                 uint k,
                                 double* arrayCurr,
                                 double* arrayPrev) {
  atomicAdd(&c, long(1));
  long indexC = index(i, j, k);

  return 2 * arrayCurr[indexC] - arrayPrev[indexC] +
         ht * ht *
             ((arrayCurr[index(i - 1, j, k)] - 2 * arrayCurr[indexC] +
               arrayCurr[index(i + 1, j, k)]) /
                  hx / hx +
              (arrayCurr[index(i, j - 1, k)] - 2 * arrayCurr[indexC] +
               arrayCurr[index(i, j + 1, k)]) /
                  hy / hy +
              (arrayCurr[index(i, j, k - 1)] - 2 * arrayCurr[indexC] +
               arrayCurr[index(i, j, k + 1)]) /
                  hz / hz);
}

struct i_j_k {
  int i, j, k;

  __device__ i_j_k(long offset) {
    long ij = offset / (kc + 2);
    i = ij / (jc + 2);
    j = ij % (jc + 2);
    k = offset % (kc + 2);
  }
};

struct j_k {
  int j, k;

  __device__ j_k(long offset) {
    j = offset / (kc + 2);
    k = offset % (kc + 2);
  }
};

struct i_j {
  int i, j;

  __device__ i_j(long offset) {
    i = offset / (jc + 2);
    j = offset % (jc + 2);
  }
};

struct i_k {
  int i, k;

  __device__ i_k(long offset) {
    i = offset / (kc + 2);
    k = offset % (kc + 2);
  }
};

struct residual_functor {
  uint curr_step;
  double* arrayNext;

  residual_functor(uint step, double* array)
      : curr_step(step), arrayNext(array) {}

  __device__ double operator()(long offset) {
    i_j_k index(offset);
    if (index.i == 0 || index.i == ic + 1 || index.j == 0 ||
        index.j == jc + 1 || index.k == 0 || index.k == kc + 1)
      return 0.0;

    double aSol =
        u(x(index.i - 1), y(index.j - 1), z(index.k - 1), deltaTime(curr_step));
    double residual = aSol - arrayNext[offset];
    return std::abs(residual);
  }
};

__host__ double cuda_residual(uint curr_step, dvector& arrayNext, long size) {
  dvector resVec(arrayNext.size());
  thrust::counting_iterator<int> it(0);
  thrust::transform(it, it + resVec.size(), resVec.begin(),
                    residual_functor(curr_step, arrayNext.data().get()));
  return thrust::reduce(resVec.begin(), resVec.end()) / size;
}

struct initPrev_functor {
  __device__ double operator()(long offset) {
    i_j_k index(offset);
    return phi(x(index.i - 1), y(index.j - 1), z(index.k - 1));
  }
};

__host__ void cuda_initPrev(dvector& arrayPrev) {
  thrust::counting_iterator<int> it(0);
  thrust::transform(it, it + arrayPrev.size(), arrayPrev.begin(),
                    initPrev_functor());
}

struct initCurr_functor {
  double* arrayPrev;

  __host__ initCurr_functor(double* array) : arrayPrev(array) {}

  __device__ double operator()(long offset) {
    i_j_k index(offset);
    return arrayPrev[offset] +
           ht * ht / 2 * (-phi(x(index.i - 1), y(index.j - 1), z(index.k - 1)));
  }
};

__host__ void cuda_initCurr(dvector& arrayPrev, dvector& arrayCurr) {
  thrust::counting_iterator<int> it(0);
  thrust::transform(it, it + arrayCurr.size(), arrayCurr.begin(),
                    initCurr_functor(arrayPrev.data().get()));
}

struct calculateIndex_functor {
  double* arrayCurr;
  double* arrayPrev;
  double* arrayNext;

  __host__ calculateIndex_functor(double* array, double* arrayP, double* arrayN)
      : arrayCurr(array), arrayPrev(arrayP), arrayNext(arrayN) {}

  __device__ double operator()(long offset) {
    i_j_k idx(offset);
    if (idx.i > 1 && idx.i < ic - 2 && idx.j > 1 && idx.j < jc - 2 &&
        idx.k > 1 && idx.k < kc - 2)
      return calculateIndex(idx.i, idx.j, idx.k, arrayCurr, arrayPrev);
    else
      return arrayNext[offset];
  }
};

__device__ bool is_border(i_j_k idx) {
  if (idx.i == 0 || idx.i == ic - 1 || idx.j == 0 || idx.j == jc - 1 ||
      idx.k == 0 || idx.k == kc - 1)
    return false;
  return (idx.i == 1 || idx.i == ic - 2 || idx.j == 1 || idx.j == jc - 2 ||
          idx.k == 1 || idx.k == kc - 2);
}

struct calculateIndexDir_functor {
  double* arrayCurr;
  double* arrayPrev;
  double* arrayNext;

  __host__ calculateIndexDir_functor(double* array,
                                     double* arrayP,
                                     double* arrayN)
      : arrayCurr(array), arrayPrev(arrayP), arrayNext(arrayN) {}

  __device__ double operator()(long offset) {
    i_j_k idx(offset);
    if (is_border(idx))
      return calculateIndex(idx.i, idx.j, idx.k, arrayCurr, arrayPrev);
    return arrayNext[offset];
  }
};

__host__ void cuda_calculateIndex(dvector& d_arrayNext,
                                  dvector& d_arrayCurr,
                                  dvector& d_arrayPrev) {
  thrust::counting_iterator<int> it(0);
  thrust::transform(
      it, it + d_arrayNext.size(), d_arrayNext.begin(),
      calculateIndex_functor(d_arrayCurr.data().get(), d_arrayPrev.data().get(),
                             d_arrayNext.data().get()));
}

void cuda_calculateDir(dvector& d_arrayNext,
                       dvector& d_arrayCurr,
                       dvector& d_arrayPrev) {
  thrust::counting_iterator<int> it(0);
  thrust::transform(it, it + d_arrayNext.size(), d_arrayNext.begin(),
                    calculateIndexDir_functor(d_arrayCurr.data().get(),
                                              d_arrayPrev.data().get(),
                                              d_arrayNext.data().get()));
}

struct edgeX_send_functor {
  int fix_i;
  double* arrayCurr;

  __host__ edgeX_send_functor(int _i, double* array)
      : fix_i(_i), arrayCurr(array) {}

  __device__ double operator()(long offset) {
    j_k idx(offset);
    return arrayCurr[index(fix_i, idx.j, idx.k)];
  }
};

struct edgeY_send_functor {
  int fix_j;
  double* arrayCurr;

  __host__ edgeY_send_functor(int j, double* array)
      : fix_j(j), arrayCurr(array) {}

  __device__ double operator()(long offset) {
    i_k idx(offset);
    return arrayCurr[index(idx.i, fix_j, idx.k)];
  }
};

struct edgeZ_send_functor {
  int fix_k;
  double* arrayCurr;

  __host__ edgeZ_send_functor(int k, double* array)
      : fix_k(k), arrayCurr(array) {}

  __device__ double operator()(long offset) {
    i_j idx(offset);
    return arrayCurr[index(idx.i, idx.j, fix_k)];
  }
};

struct edgeX_recv_functor {
  int fix_i;
  double* v;
  double* arrayCurr;

  __host__ edgeX_recv_functor(int _i, double* array, double* arrayC)
      : fix_i(_i), v(array), arrayCurr(arrayC) {}

  __device__ double operator()(long offset) {
    i_j_k idx(offset);
    if (idx.i == fix_i)
      return v[index2(idx.j, idx.k)];
    else
      return arrayCurr[offset];
  }
};

struct edgeY_recv_functor {
  int fix_j;
  double* v;
  double* def;

  __host__ edgeY_recv_functor(int _j, double* array, double* d)
      : fix_j(_j), v(array), def(d) {}

  __device__ double operator()(long offset) {
    i_j_k idx(offset);
    if (idx.j == fix_j)
      return v[index2(idx.i, idx.k)];
    else
      return def[offset];
  }
};

struct edgeZ_recv_functor {
  int fix_k;
  double* v;
  double* def;

  __host__ edgeZ_recv_functor(int _k, double* array, double* d)
      : fix_k(_k), v(array), def(d) {}

  __device__ double operator()(long offset) {
    i_j_k idx(offset);
    if (idx.k == fix_k)
      return v[index2(idx.i, idx.j)];
    else
      return def[offset];
  }
};

void cuda_edgeX(ExchangeDir cdir,
                dvector& v,
                uint id,
                bool recv,
                dvector& d_arrayNext,
                dvector& d_arrayCurr,
                long ic,
                long jc,
                long kc) {
  int i;
  switch (cdir) {
    case plus_x: {
      i = recv ? ic - 1 : ic - 2;
      break;
    }
    case minus_x: {
      i = recv ? 0 : 1;
      break;
    }
    case period_plus_x: {
      i = recv ? ic - 1 : ic - 2;
      break;
    }
    case period_minus_x: {
      i = recv ? 1 : 2;
      break;
    }
    default:
      std::cerr << "BAD CASE" << std::endl;
      exit(1);
  }

  thrust::counting_iterator<int> it(0);

  if (!recv) {
    thrust::transform(it, it + v.size(), v.begin(),
                      edgeX_send_functor(i, d_arrayCurr.data().get()));
  } else {
    thrust::transform(it, it + v.size(), v.begin(),
                      edgeX_recv_functor(i, d_arrayCurr.data().get(),
                                         d_arrayCurr.data().get()));
  }
}

void cuda_edgeY(ExchangeDir cdir,
                dvector& v,
                uint id,
                bool recv,
                dvector& d_arrayNext,
                dvector& d_arrayCurr,
                long jc) {
  int j;
  switch (cdir) {
    case plus_y: {
      j = recv ? jc - 1 : jc - 2;
      break;
    }
    case minus_y: {
      j = recv ? 0 : 1;
      break;
    }
    default:
      std::cerr << "BAD CASE" << std::endl;
      exit(1);
  }

  thrust::counting_iterator<int> it(0);

  if (!recv) {
    thrust::transform(it, it + v.size(), v.begin(),
                      edgeY_send_functor(j, d_arrayCurr.data().get()));
  } else {
    thrust::transform(it, it + v.size(), v.begin(),
                      edgeY_recv_functor(j, d_arrayCurr.data().get(),
                                         d_arrayCurr.data().get()));
  }
}

void cuda_edgeZ(ExchangeDir cdir,
                dvector& v,
                uint id,
                bool recv,
                dvector& d_arrayNext,
                dvector& d_arrayCurr,
                long ic,
                long jc,
                long kc) {
  int k;
  switch (cdir) {
    case plus_z: {
      k = recv ? kc - 1 : kc - 2;
      break;
    }
    case minus_z: {
      k = recv ? 0 : 1;
      break;
    }
    case period_plus_z: {
      k = recv ? kc - 1 : kc - 2;
      break;
    }
    case period_minus_z: {
      k = recv ? 1 : 2;
      break;
    }
    default:
      std::cerr << "BAD CASE" << std::endl;
      exit(1);
  }
  thrust::counting_iterator<int> it(0);

  if (!recv) {
    thrust::transform(it, it + v.size(), v.begin(),
                      edgeZ_send_functor(k, d_arrayCurr.data().get()));
  } else {
    thrust::transform(it, it + v.size(), v.begin(),
                      edgeZ_recv_functor(k, d_arrayCurr.data().get(),
                                         d_arrayCurr.data().get()));
  }
}
